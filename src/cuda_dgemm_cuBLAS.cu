#include <chrono>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <vector>

#include "../include/common_utils.hpp"

int main() {
    int mat_sizes[] = {128, 256, 512, 1024, 2048, 4096, 8192, 16384};
    int n_sizes = mat_sizes.size();

    // Loop over all Matrix sizes
    for (int i = 0; i < n_sizes; i++) {
        int N = mat_sizes[i];
        int size = N * N;

        // Initialize memory on host
        std::vector<double> h_A(size), h_B(size), h_C(size);
        random_matrix<double>(h_A.data(), N);
        random_matrix<double>(h_B.data(), N);

        // Allocate memory on device and copy
        double *d_A, *d_B, *d_C;
        hipMalloc(&d_A, size * sizeof(double));
        hipMalloc(&d_B, size * sizeof(double));
        hipMalloc(&d_C, size * sizeof(double));

        hipMemcpy(d_A, h_A.data(), size * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B.data(), size * sizeof(double), hipMemcpyHostToDevice);

        // Create cuBLAS handle
        hipblasHandle_t handle;
        hipblasCreate(&handle);

        // Warmup
        double alpha = random_value<double>(), beta = random_value<double>();
        hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C,
                    N);
        hipDeviceSynchronize();

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Timed runs
        const int n_repeats = 10;
        hipEventRecord(start);

        for (int j = 0; j < n_repeats; j++) {
            hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_B, N, &beta,
                        d_C, N);
        }
        hipDeviceSynchronize();
        hipEventRecord(stop);

        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        double avg_time = milliseconds / n_repeats;
        double tflops = 2.0 * N * N * N * 1e-9 / avg_time;

        std::cout << "N: " << std::setw(6) << N << " | Time: " << std::fixed << std::setprecision(6)
                  << avg_time << " s"
                  << " | TFLOPS: " << std::fixed << std::setprecision(2) << tflops << "\n";

        // Cleanup
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        hipblasDestroy(handle);
    }

    return 0;
}
