#include <chrono>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <vector>

#include "../include/common_utils.hpp"

int main() {
    int device_id;

    // Get the current device being used
    hipGetDevice(&device_id);

    std::cout << "Running on GPU: " << device_id << std::endl;

    int mat_sizes[] = {128, 256, 512, 1024, 2048, 4096, 8192, 16384};
    int n_sizes = sizeof(mat_sizes) / sizeof(mat_sizes[0]);

    // Loop over all Matrix sizes
    for (int i = 0; i < n_sizes; i++) {
        int N = mat_sizes[i];
        int size = N * N;

        // Initialize memory on host
        std::vector<double> h_A(size), h_B(size), h_C(size);
        random_matrix<double>(h_A.data(), N);
        random_matrix<double>(h_B.data(), N);

        // Allocate memory on device and copy
        double *d_A, *d_B, *d_C;
        hipMalloc(&d_A, size * sizeof(double));
        hipMalloc(&d_B, size * sizeof(double));
        hipMalloc(&d_C, size * sizeof(double));

        hipMemcpy(d_A, h_A.data(), size * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B.data(), size * sizeof(double), hipMemcpyHostToDevice);

        // Create cuBLAS handle
        hipblasHandle_t handle;
        hipblasCreate(&handle);

        // Warmup
        double alpha = random_value<double>(), beta = random_value<double>();
        hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C,
                    N);
        hipDeviceSynchronize();

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Timed runs
        const int n_repeats = 10;
        hipEventRecord(start);

        for (int j = 0; j < n_repeats; j++) {
            hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_B, N, &beta,
                        d_C, N);
        }
        hipDeviceSynchronize();
        hipEventRecord(stop);

        // Kerneltime calculation
        hipEventSynchronize(stop);
        float elapsedTime = 0;
        hipEventElapsedTime(&elapsedTime, start, stop);

        double avg_time = (elapsedTime / 1000) / n_repeats;
        double gflops = 2.0 * N * N * N * 1e-9 / avg_time;

        std::cout << "N: " << std::setw(6) << N << " | Time: " << std::fixed << std::setprecision(6)
                  << avg_time << " s"
                  << " | GFLOPS: " << std::fixed << std::setprecision(2) << gflops << "\n";

        // Cleanup
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        hipblasDestroy(handle);
    }

    return 0;
}
